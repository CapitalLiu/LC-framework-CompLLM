#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>


typedef float type;  // type of a word

static const int TPB = 512;  // threads per block

static const type missing = nanf("");


static __device__ inline bool isMissing(const type x)
{
  return isnan(x);
}


static __device__ inline void reconstruct(type& a, type& b)
{
  if (isMissing(b)) {
    b = a;  // replace missing value by average
  } else {
    a = 2 * a - b;  // replace by 2 * average minus given value
  }
}


static __global__ void unshuffle(const int iter, const int step, const int shift1, const int len, const type* const __restrict__ input, type* const __restrict__ output)
{
  const int i = threadIdx.x + blockIdx.x * TPB;
  if (i < iter) {
    const int ishift = i << step;
    const int offs = ishift + shift1;
    output[offs] = input[len + i];
  }
  if ((len == 1) && (i == 0)) {
    output[0] = input[0];
  }
}


static __global__ void decode(const int iter, const int step, const int shift1, type* const __restrict__ output)
{
  const int i = threadIdx.x + blockIdx.x * TPB;
  if (i < iter) {
    const int ishift = i << step;
    const int offs = ishift + shift1;
    reconstruct(output[ishift], output[offs]);
  }
}


static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}


struct GPUTimer
{
  hipEvent_t beg, end;
  GPUTimer() {hipEventCreate(&beg);  hipEventCreate(&end);}
  ~GPUTimer() {hipEventDestroy(beg);  hipEventDestroy(end);}
  void start() {hipEventRecord(beg, 0);}
  double stop() {hipEventRecord(end, 0);  hipEventSynchronize(end);  float ms;  hipEventElapsedTime(&ms, beg, end);  return 0.001 * ms;}
};


int main(int argc, char* argv[])
{
  printf("Progressive decoder v0.1 CUDA (%s)\n", __FILE__);
  if (argc != 3) {printf("USAGE: %s input_file_name output_file_name\n", argv[0]);  exit(-1);}

  // read input from file
  FILE* const fin = fopen(argv[1], "rb");  assert(fin != NULL);
  fseek(fin, 0, SEEK_END);  assert(ftell(fin) >= sizeof(int) + sizeof(type));
  fseek(fin, 0, SEEK_SET);
  int size;
  fread(&size, sizeof(int), 1, fin);
  printf("size: %d words, %d bytes\n", size, size * sizeof(type));
  type* const input = new type [size];
  const int insize = fread(input, sizeof(type), size, fin);
  fclose(fin);
  printf("input size: %d words, %d bytes\n", insize, insize * sizeof(type));
  for (int i = insize; i < size; i++) input[i] = missing;
  if (size < 1) {fprintf(stderr, "ERROR: input must contain at least one value\n\n");  exit(-1);}

  // check GPU
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {fprintf(stderr, "ERROR: there is no CUDA capable device\n\n");  exit(-1);}
  const int SMs = deviceProp.multiProcessorCount;
  const int mTSM = deviceProp.maxThreadsPerMultiProcessor;
  printf("gpu: %s with %d SMs and %d mTpSM (%.1f MHz and %.1f MHz)\n", deviceProp.name, SMs, mTSM, deviceProp.clockRate * 0.001, deviceProp.memoryClockRate * 0.001);

  // alloc GPU memory
  type* d_input;
  hipMalloc((void **)&d_input, size * sizeof(type));
  hipMemcpy(d_input, input, size * sizeof(type), hipMemcpyHostToDevice);
  type* d_output;
  hipMalloc((void **)&d_output, size * sizeof(type));
  CheckCuda();

  // start timer
  GPUTimer timer;
  timer.start();

  // compute sizes
  int s [60];
  int k [60];
  int level = 0;
  int len = size;
  do {
    k[level] = len / 2;
    len = (len + 1) / 2;
    s[level] = len;
    level++;
  } while (len > 1);

  // launch kernels
  for (int lvl = level - 1; lvl >= 0; lvl--) {
    const int len = s[lvl];
    const int iter = k[lvl];
    const int shift1 = 1 << lvl;
    const int step = lvl + 1;
    unshuffle<<<(iter + TPB - 1) / TPB, TPB>>>(iter, step, shift1, len, d_input, d_output);
  }
  for (int lvl = level - 1; lvl >= 0; lvl--) {
    const int iter = k[lvl];
    const int shift1 = 1 << lvl;
    const int step = lvl + 1;
    decode<<<(iter + TPB - 1) / TPB, TPB>>>(iter, step, shift1, d_output);
  }

  // stop timer
  hipDeviceSynchronize();
  double runtime = timer.stop();
  CheckCuda();
  printf("compute time: %.6f s\n", runtime);
  printf("throughput: %.3f Gbytes/s\n", size * sizeof(type) * 0.000000001 / runtime);

  // get result
  type* const output = new type [size];
  hipMemcpy(output, d_output, size * sizeof(type), hipMemcpyDeviceToHost);
  CheckCuda();

  // write output to file
  FILE* const fout = fopen(argv[2], "wb");  assert(fout != NULL);
  fwrite(output, sizeof(type), size, fout);
  fclose(fout);

  // clean up
  delete [] input;
  delete [] output;
  hipFree(d_input);
  hipFree(d_output);
  return 0;
}
